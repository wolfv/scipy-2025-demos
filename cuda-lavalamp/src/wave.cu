#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <cstdio>
#include <ctime>

#ifdef _WIN32
#include <windows.h>
#define SLEEP_MS(x) Sleep(x)
#define IS_WINDOWS 1
#else
#include <unistd.h>
#define SLEEP_MS(x) usleep((x)*1000)
#define IS_WINDOWS 0
#endif

#define WIDTH 60
#define HEIGHT 30
#define GRAVITY -0.005f
#define THERMAL_FORCE 0.10f
#define VISCOSITY 0.96f
#define COOLING_RATE 0.998f
#define HEATING_RATE 0.04f

struct Particle {
    float x, y;
    float vx, vy;
    float temperature;
    float density;
    bool active;
};

__device__ float simpleRandom(unsigned int* seed) {
    *seed = (*seed * 1664525 + 1013904223);
    return (float)(*seed) / 4294967296.0f;
}

__global__ void initParticles(Particle* particles, int numParticles, unsigned int baseSeed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numParticles) return;

    unsigned int seed = baseSeed + idx * 7919;

    // Start particles randomly distributed throughout the lamp
    particles[idx].x = simpleRandom(&seed) * (WIDTH - 6) + 3;
    particles[idx].y = simpleRandom(&seed) * (HEIGHT - 4) + 2;
    particles[idx].vx = (simpleRandom(&seed) - 0.5f) * 0.02f;
    particles[idx].vy = (simpleRandom(&seed) - 0.5f) * 0.02f;
    particles[idx].temperature = simpleRandom(&seed) * 0.6f + 0.2f;
    particles[idx].density = 1.0f - particles[idx].temperature * 0.4f; // Hot = less dense
    particles[idx].active = true;
}

__global__ void updateParticles(Particle* particles, int numParticles, int width, int height, int frame) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numParticles) return;

    Particle* p = &particles[idx];
    unsigned int seed = frame * 31415 + idx * 2718;

    // Heat source at bottom (like the lamp's heating element)
    float distFromBottom = p->y / (float)height;
    if (distFromBottom < 0.15f) {
        // Strong heating near bottom
        p->temperature += HEATING_RATE * (0.15f - distFromBottom) * 6.0f;
        if (p->temperature > 1.0f) p->temperature = 1.0f;
    }

    // Cooling at top (like touching the cool top of the lamp)
    if (distFromBottom > 0.85f) {
        p->temperature *= 0.995f; // Faster cooling at top
    } else {
        // Gradual cooling throughout the lamp
        p->temperature *= COOLING_RATE;
    }

    // Update density based on temperature (hot = less dense = rises)
    p->density = 1.2f - p->temperature * 0.5f;

    // Buoyancy force - hot particles rise, cool particles sink
    float buoyancy = (0.8f - p->density) * THERMAL_FORCE;
    p->vy += GRAVITY + buoyancy;

    // Add some convection currents and turbulence
    float convection = sinf(p->x * 0.1f + frame * 0.01f) * 0.001f;
    p->vx += convection;
    p->vx += (simpleRandom(&seed) - 0.5f) * 0.002f;
    p->vy += (simpleRandom(&seed) - 0.5f) * 0.001f;

    // Apply viscosity (lava is thick!)
    p->vx *= VISCOSITY;
    p->vy *= VISCOSITY;

    // Update position
    p->x += p->vx;
    p->y += p->vy;

    // Boundary conditions - bounce off walls
    if (p->x <= 2) { 
        p->x = 2; 
        p->vx = -p->vx * 0.5f; 
    }
    if (p->x >= width - 3) { 
        p->x = width - 3; 
        p->vx = -p->vx * 0.5f; 
    }

    // Bounce off top and bottom with some energy loss
    if (p->y <= 1) { 
        p->y = 1; 
        p->vy = -p->vy * 0.3f;
        // Extra heating when hitting the bottom heating element
        p->temperature += 0.05f;
        if (p->temperature > 1.0f) p->temperature = 1.0f;
    }
    if (p->y >= height - 2) { 
        p->y = height - 2; 
        p->vy = -p->vy * 0.3f;
        // Extra cooling when hitting the top
        p->temperature *= 0.95f;
    }

    // Ensure temperature bounds
    if (p->temperature < 0.0f) p->temperature = 0.0f;
    if (p->temperature > 1.0f) p->temperature = 1.0f;
}

__global__ void renderField(Particle* particles, int numParticles, float* field, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    int idx = y * width + x;
    field[idx] = 0.0f;

    // Add base temperature gradient (hot at bottom, cool at top)
    float baseTemp = expf(-((float)y / height) * 2.5f) * 0.25f;
    field[idx] += baseTemp;

    // Add particle contributions
    for (int i = 0; i < numParticles; i++) {
        float dx = (float)x - particles[i].x;
        float dy = (float)y - particles[i].y;
        float dist = sqrtf(dx * dx + dy * dy);

        if (dist < 4.0f) {
            float influence = expf(-dist * dist * 0.3f);
            field[idx] += particles[i].temperature * influence * 0.7f;
        }
    }

    if (field[idx] > 1.0f) field[idx] = 1.0f;
}

void clearScreen() {
#ifdef _WIN32
    system("cls");
#else
    printf("\033[2J\033[H");
#endif
}

void renderLavaLamp(float* field, int width, int height, int frame) {
    clearScreen();

    // Draw lamp container
    if (IS_WINDOWS) {
        printf("    +");
        for (int i = 0; i < width; i++) printf("-");
        printf("+\n");
    } else {
        printf("    ╔");
        for (int i = 0; i < width; i++) printf("═");
        printf("╗\n");
    }

    for (int y = height - 1; y >= 0; y--) {
        printf("    %s", IS_WINDOWS ? "|" : "║");
        for (int x = 0; x < width; x++) {
            float val = field[y * width + x];

            if (IS_WINDOWS) {
                char c;
                if (val > 0.8f) c = '#';        // Very hot
                else if (val > 0.6f) c = '@';   // Hot
                else if (val > 0.4f) c = '*';   // Warm
                else if (val > 0.25f) c = 'o';  // Lukewarm
                else if (val > 0.15f) c = '.';  // Cool
                else if (val > 0.05f) c = '\''; // Cold
                else c = ' ';                   // Background
                printf("%c", c);
            } else {
                // Use colors for better effect on Unix terminals
                if (val > 0.8f) printf("\033[91m█\033[0m");      // Bright red
                else if (val > 0.6f) printf("\033[93m▓\033[0m"); // Yellow
                else if (val > 0.4f) printf("\033[33m▒\033[0m"); // Orange
                else if (val > 0.25f) printf("\033[31m░\033[0m"); // Red
                else if (val > 0.15f) printf("\033[90m·\033[0m"); // Gray
                else if (val > 0.05f) printf("\033[34m'\033[0m"); // Blue (cool)
                else printf(" ");
            }
        }
        printf("%s\n", IS_WINDOWS ? "|" : "║");
    }

    if (IS_WINDOWS) {
        printf("    +");
        for (int i = 0; i < width; i++) printf("-");
        printf("+\n");
        printf("    [========== HEATING ELEMENT ==========]\n");
        printf("    LAVA LAMP SIMULATION - Frame %d\n", frame);
    } else {
        printf("    ╚");
        for (int i = 0; i < width; i++) printf("═");
        printf("╝\n");
        printf("    🔥🔥🔥🔥🔥 HEATING ELEMENT 🔥🔥🔥🔥🔥\n");
        printf("    🌋 LAVA LAMP SIMULATION - Frame %d 🌋\n", frame);
    }

    fflush(stdout);
}

int main() {
    const int numParticles = 180;
    const int fieldSize = WIDTH * HEIGHT * sizeof(float);

#ifdef _WIN32
    SetConsoleOutputCP(CP_UTF8);
    SetConsoleCP(CP_UTF8);
#endif

    printf("Initializing CUDA...\n");

    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        printf("No CUDA devices found!\n");
        return -1;
    }

    float* h_field = (float*)malloc(fieldSize);
    Particle* d_particles;
    float* d_field;

    hipMalloc(&d_particles, numParticles * sizeof(Particle));
    hipMalloc(&d_field, fieldSize);

    hipMemset(d_field, 0, fieldSize);

    dim3 particleBlock(256);
    dim3 particleGrid((numParticles + particleBlock.x - 1) / particleBlock.x);

    dim3 fieldBlock(16, 16);
    dim3 fieldGrid((WIDTH + fieldBlock.x - 1) / fieldBlock.x, 
                   (HEIGHT + fieldBlock.y - 1) / fieldBlock.y);

    printf("Initializing lava particles...\n");
    unsigned int seed = (unsigned int)time(NULL);
    initParticles<<<particleGrid, particleBlock>>>(d_particles, numParticles, seed);
    hipDeviceSynchronize();

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        return -1;
    }

    printf("🌋 Starting Lava Lamp Simulation...\n");
    printf("Watch the hot particles rise and cool particles sink!\n");
    if (IS_WINDOWS) {
        printf("Note: For best visuals, use Windows Terminal\n");
    }
    SLEEP_MS(2000);

    for (int frame = 0; frame < 10000; frame++) {
        // Update particle physics
        updateParticles<<<particleGrid, particleBlock>>>(d_particles, numParticles, WIDTH, HEIGHT, frame);

        // Render temperature field
        renderField<<<fieldGrid, fieldBlock>>>(d_particles, numParticles, d_field, WIDTH, HEIGHT);

        // Display every few frames
        if (frame % 3 == 0) {
            hipMemcpy(h_field, d_field, fieldSize, hipMemcpyDeviceToHost);
            renderLavaLamp(h_field, WIDTH, HEIGHT, frame);
            SLEEP_MS(120);
        }

        // Occasionally add some thermal disturbance
        if (frame % 300 == 0) {
            initParticles<<<1, 20>>>(d_particles, 20, seed + frame);
        }
    }

    free(h_field);
    hipFree(d_particles);
    hipFree(d_field);

    return 0;
}
